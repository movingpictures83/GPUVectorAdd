#include <emmintrin.h>
#include <sys/time.h> 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>
#include <iomanip>
#include <fstream>

#include "GPUVectorAddPlugin.h"

void GPUVectorAddPlugin::input(std::string file) {
 inputfile = file;
 readParameterFile(file);
 N = atoi(myParameters["N"].c_str());
 A = (float*) malloc(N*sizeof(float));
 B = (float*) malloc(N*sizeof(float));
 C = (float*) malloc(N*sizeof(float));
 std::ifstream myinput((std::string(PluginManager::prefix())+myParameters["vector1"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < N; ++i) {
	float k;
	myinput >> k;
        A[i] = k;
 }
 std::ifstream myinput2((std::string(PluginManager::prefix())+myParameters["vector2"]).c_str(), std::ios::in);
 for (i = 0; i < N; ++i) {
	float k;
	myinput2 >> k;
        B[i] = k;
 }
}




void GPUVectorAddPlugin::run() {
	float *pA;
	float *pB;
	float *pC;
hipMalloc((void**)&pA, (N)*sizeof(float));
hipMalloc((void**)&pB, (N)*sizeof(float));
hipMalloc((void**)&pC, (N)*sizeof(float));
hipMemcpy(pA, A, (N)*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (N)*sizeof(float), hipMemcpyHostToDevice);
printf("***Add on %d x %d Matrix on GPU***\n",N,N);
vecAdd<<<1,N>>>(pA, pB, pC, N);
hipMemcpy(C, pC, (N)*sizeof(float), hipMemcpyDeviceToHost);

hipFree(pA);
hipFree(pB);
hipFree(pC);

}

void GPUVectorAddPlugin::output(std::string file) {
	std::ofstream outfile(file.c_str(), std::ios::out);
	int i, j;
        for (i = 0; i < N; ++i){
		outfile << C[i];//std::setprecision(0) << a[i*N+j];
		outfile << "\n";
	}
	free(A);
	free(B);
	free(C);
}



PluginProxy<GPUVectorAddPlugin> GPUVectorAddPluginProxy = PluginProxy<GPUVectorAddPlugin>("GPUVectorAdd", PluginManager::getInstance());


